#include "hip/hip_runtime.h"
#include "CudaCooleyTukeyFFT.cuh"
#include "../CudaCommon/CudaCommon.cuh"

namespace fftcore::cudakernels
{
    template <typename FloatingType>
    __global__ void d_butterfly_kernel_cooleytukey(ComplexCuda<FloatingType> * __restrict__ input_output, unsigned m2)
    {
        using ComplexCuda = ComplexCuda<FloatingType>;

        unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
        unsigned int k, j;
        ComplexCuda w, t, u;

        if (tid < d_n2){

            j = tid & (m2 - 1);  // j = tid % m2
            k = (tid >> (__ffs(m2) - 1)) * m2 * 2;  // k = (tid / m2) * m

            // w = exp(ComplexCuda(0, d_fft_sign * M_PI * j / m2));
            w = ComplexCuda(__cosf(d_fft_sign * M_PI * j / m2), __sinf(d_fft_sign * M_PI * j / m2));
            t = w * input_output[k + j + m2];
            u = input_output[k + j];

            input_output[k + j] = u + t;
            input_output[k + j + m2] = u - t;
        }
    }
}

namespace fftcore
{   

    using cudautils::gpuAssert;

    template <typename FloatingType>
    CudaCooleyTukeyFFT<FloatingType>::CudaCooleyTukeyFFT()
    {
        gpuErrchk( hipFree(0) ); //initialize CUDA context
    }

    template <typename FloatingType>
    void CudaCooleyTukeyFFT<FloatingType>::fft(const CTensor_1D &input, CTensor_1D &output, FFTDirection fftDirection) const
    {
        output = input; //deep copy
        fft(output, fftDirection);
    };

    template <typename FloatingType>
    void CudaCooleyTukeyFFT<FloatingType>::fft(const RTensor_1D &, CTensor_1D &, FFTDirection) const
    {
        throw NotSupportedException("Operation is not supported");
    };

    template <typename FloatingType>
    void CudaCooleyTukeyFFT<FloatingType>::fft(CTensor_1D &input_output, fftcore::FFTDirection fftDirection) const
        {
        
        using cudakernels::d_butterfly_kernel_cooleytukey;
        using cudakernels::d_bit_reversal_permutation;
        using cudakernels::d_scale;
        using cudakernels::d_fft_sign, cudakernels::d_n2;
        using ComplexCuda = cudakernels::ComplexCuda<FloatingType>;

        const TensorIdx n = input_output.size(), n2 = n / 2, log2n = std::log2(n);

        //allocate memory on device
        ComplexCuda *d_input_output;
        gpuErrchk( hipMalloc((void **)&d_input_output, n * sizeof(ComplexCuda)) );
            
        //copy input to device
        gpuErrchk( hipMemcpy(d_input_output, input_output.data(), n * sizeof(ComplexCuda), hipMemcpyHostToDevice) );

        //bit reversal permutation on device
        unsigned int threadsPerBlock = THREADS_PER_BLOCK;
        unsigned int numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;
        d_bit_reversal_permutation<<<numBlocks, threadsPerBlock>>>(d_input_output, n, log2n);

        //set fftDirection on device constant memory
        char sign = (fftDirection == FFT_FORWARD) ? -1 : 1;
        gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_fft_sign), &sign, sizeof(char)) );

        //set n2 on device constant memory
        gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_n2), &n2, sizeof(TensorIdx)) );

        //Cooley-Tukey iterative FFT
        numBlocks = (n2 + threadsPerBlock - 1) / threadsPerBlock; //set number of blocks so that each thread will process 2 elements
        TensorIdx m, m2;
        for(unsigned int s = 1; s <= log2n; ++s){

            m = 1 << s;  // 2^s
            m2 = m >> 1; // m2 = m/2
            d_butterfly_kernel_cooleytukey<<<numBlocks, threadsPerBlock>>>(d_input_output, m2);

        }

        //scale output if inverse FFT
        if(fftDirection == FFT_INVERSE){
            numBlocks = (n + threadsPerBlock - 1) / threadsPerBlock;
            d_scale<<<numBlocks, threadsPerBlock>>>(d_input_output, n);
        }

        //copy output to host
        gpuErrchk( hipMemcpy(input_output.data(), d_input_output, n * sizeof(ComplexCuda), hipMemcpyDeviceToHost) );

        //free memory on device
        gpuErrchk( hipFree(d_input_output) );
    };

} // namespace fftcore

namespace fftcore
{
    template class CudaCooleyTukeyFFT<float>;
    template class CudaCooleyTukeyFFT<double>;
} // namespace fftcore